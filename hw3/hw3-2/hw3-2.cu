
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

const int INF = ((1 << 30) - 1);
void input(char* inFileName);
void output(char* outFileName);

void block_FW();
int ceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;
int padN;
int* Dist;

int main(int argc, char* argv[]) {
    input(argv[1]);
    block_FW();
    output(argv[2]);
    return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    printf("vertice = %d\n", n);
    int pad = n % 32;
    padN = n + (32-pad);
    Dist = (int*) malloc(sizeof(int)*padN*padN);
    for (int i = 0; i < padN; ++i) {
        for (int j = 0; j < padN; ++j) {
            if (i == j) {
                Dist[i*padN+j] = 0;
            } else {
                Dist[i*padN+j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*padN+pair[1]]= pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i*padN+j] >= INF) Dist[i*padN+j] = INF;
        }
        fwrite(&Dist[i*padN], sizeof(int), n, outfile);
    }
    fclose(outfile);
}

__global__ void blockedASAPPhase1(int round, int* graph, int V, int blockSize) {
    __shared__ int shareDistInBlock[32*32];
    int getBlockSize = blockSize;
    int i = blockSize * round + threadIdx.y;
    int j = blockSize * round + threadIdx.x;
    shareDistInBlock[threadIdx.y * getBlockSize + threadIdx.x] = graph[i*V+j];
    __syncthreads();
    for(int k = 0; k<getBlockSize; k++) {
        shareDistInBlock[threadIdx.y * getBlockSize + threadIdx.x] = min(shareDistInBlock[threadIdx.y * getBlockSize + threadIdx.x], shareDistInBlock[threadIdx.y*getBlockSize + k] + shareDistInBlock[k*getBlockSize + threadIdx.x]);
        // if(shareDistInBlock[threadIdx.x * getBlockSize + threadIdx.y] > shareDistInBlock[threadIdx.x*getBlockSize + k] + shareDistInBlock[k*getBlockSize + threadIdx.y]){
        //     shareDistInBlock[threadIdx.x * getBlockSize + threadIdx.y] = shareDistInBlock[threadIdx.x*getBlockSize + k] + shareDistInBlock[k*getBlockSize + threadIdx.y];
        // }
        __syncthreads();
    }
    graph[i*V+j] = shareDistInBlock[threadIdx.y*getBlockSize+threadIdx.x];
}

// __global__ void blockedASAPPhase2Row(int round, int* graph, int V, int blockSize) {
//     if(blockIdx.x != round) {
//         __shared__ int shareDistInBlockPivot[32*32];
//         __shared__ int shareDistInBlockOther[32*32];
//         int getBlockSize = blockSize;
//         int pivotI = getBlockSize * round + threadIdx.x;
//         int pivotJ = getBlockSize * round + threadIdx.y;
//         int targetI = pivotI;
//         int targetJ = blockIdx.x * getBlockSize + threadIdx.y;

//         if(pivotI < V && pivotJ < V) {
//             shareDistInBlockPivot[threadIdx.x * getBlockSize + threadIdx.y] = graph[pivotI*V+pivotJ];
//         }
//         else {
//             shareDistInBlockPivot[threadIdx.x * getBlockSize + threadIdx.y] = INF;
//         }

//         if(targetI < V && targetJ < V) {
//             shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = graph[targetI*V+targetJ];
//         }
//         else {
//             shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = INF;
//         }
//         __syncthreads();
//         if(targetI < V && targetJ < V) {
//             for(int k=0; k<getBlockSize; k++) {
//                 if(shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] > shareDistInBlockPivot[threadIdx.x*getBlockSize + k] + shareDistInBlockOther[k*getBlockSize + threadIdx.y]) {
//                     shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = shareDistInBlockPivot[threadIdx.x*getBlockSize + k] + shareDistInBlockOther[k*getBlockSize + threadIdx.y];
//                 }
//             }
//             graph[targetI*V+targetJ] = shareDistInBlockOther[threadIdx.x*getBlockSize+threadIdx.y];
//         }
//     }
// }

// __global__ void blockedASAPPhase2Col(int round, int* graph, int V, int blockSize) {
//     if(blockIdx.x != round) {
//         __shared__ int shareDistInBlockPivot[32*32];
//         __shared__ int shareDistInBlockOther[32*32];
//         int getBlockSize = blockSize;
//         int pivotI = getBlockSize * round + threadIdx.x;
//         int pivotJ = getBlockSize * round + threadIdx.y;
//         int targetI = blockIdx.x * getBlockSize + threadIdx.x;
//         int targetJ = pivotJ;

//         if(pivotI < V && pivotJ < V) {
//             shareDistInBlockPivot[threadIdx.x * getBlockSize + threadIdx.y] = graph[pivotI*V+pivotJ];
//         }
//         else {
//             shareDistInBlockPivot[threadIdx.x * getBlockSize + threadIdx.y] = INF;
//         }

//         if(targetI < V && targetJ < V) {
//             shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = graph[targetI*V+targetJ];
//         }
//         else {
//             shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = INF;
//         }
//         __syncthreads();
//         if(targetI < V && targetJ < V) {
//             for(int k=0; k<getBlockSize; k++) {
//                 if(shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] > shareDistInBlockOther[threadIdx.x*getBlockSize + k] + shareDistInBlockPivot[k*getBlockSize + threadIdx.y]) {
//                     shareDistInBlockOther[threadIdx.x * getBlockSize + threadIdx.y] = shareDistInBlockOther[threadIdx.x*getBlockSize + k] + shareDistInBlockPivot[k*getBlockSize + threadIdx.y];
//                 }
//             }
//             graph[targetI*V+targetJ] = shareDistInBlockOther[threadIdx.x*getBlockSize+threadIdx.y];
//         }
//     }
// }

__global__ void blockedASAPPhase2(int round, int* graph, int V, int blockSize) {
    if(blockIdx.x != round) {
        __shared__ int shareDistInBlockPivot[32*32];
        __shared__ int shareDistInBlockOtherRow[32*32];
        __shared__ int shareDistInBlockOtherCol[32*32];
        int getBlockSize = blockSize;
        int pivotI = getBlockSize * round + threadIdx.y;
        int pivotJ = getBlockSize * round + threadIdx.x;
        int targetIRow = pivotI;
        int targetJRow = blockIdx.x * getBlockSize + threadIdx.x;
        int targetICol = blockIdx.x * getBlockSize + threadIdx.y;
        int targetJCol = pivotJ;
        shareDistInBlockPivot[threadIdx.y * getBlockSize + threadIdx.x] = graph[pivotI*V+pivotJ];
        shareDistInBlockOtherRow[threadIdx.y * getBlockSize + threadIdx.x] = graph[targetIRow*V+targetJRow];
        shareDistInBlockOtherCol[threadIdx.y * getBlockSize + threadIdx.x] = graph[targetICol*V+targetJCol];
        __syncthreads();
        for(int k=0; k<getBlockSize; k++) {
            shareDistInBlockOtherRow[threadIdx.y * getBlockSize + threadIdx.x] = min(shareDistInBlockOtherRow[threadIdx.y * getBlockSize + threadIdx.x], shareDistInBlockPivot[threadIdx.y*getBlockSize + k] + shareDistInBlockOtherRow[k*getBlockSize + threadIdx.x]);
            shareDistInBlockOtherCol[threadIdx.y * getBlockSize + threadIdx.x] = min(shareDistInBlockOtherCol[threadIdx.y * getBlockSize + threadIdx.x], shareDistInBlockOtherCol[threadIdx.y*getBlockSize + k] + shareDistInBlockPivot[k*getBlockSize + threadIdx.x]);
        }
        graph[targetIRow*V+targetJRow] = shareDistInBlockOtherRow[threadIdx.y*getBlockSize+threadIdx.x];
        graph[targetICol*V+targetJCol] = shareDistInBlockOtherCol[threadIdx.y*getBlockSize+threadIdx.x];
    }
}

__global__ void blockedASAPPhase3(int round, int* graph, int V, int blockSize) {
    if(blockIdx.x != round && blockIdx.y != round) {
        __shared__ int shareDistInBlockPivotRow[32*32];
        __shared__ int shareDistInBlockPivotCol[32*32];
        int getBlockSize = blockSize;
        int pivotRowI = getBlockSize * round + threadIdx.y;
        int pivotRowJ = blockIdx.y * blockDim.x + threadIdx.x;
        int pivotColI = blockIdx.x * blockDim.y + threadIdx.y;
        int pivotColJ = getBlockSize * round + threadIdx.x;
        int pointUpdatedByThisThread = graph[ pivotColI*V + pivotRowJ];
        shareDistInBlockPivotRow[threadIdx.y * getBlockSize + threadIdx.x] = graph[pivotRowI*V+pivotRowJ];
        shareDistInBlockPivotCol[threadIdx.y * getBlockSize + threadIdx.x] = graph[pivotColI*V+pivotColJ];
        __syncthreads();

        for(int k=0; k<getBlockSize; k++) {
            pointUpdatedByThisThread = min(pointUpdatedByThisThread, shareDistInBlockPivotCol[threadIdx.y*getBlockSize + k] + shareDistInBlockPivotRow[k*getBlockSize + threadIdx.x]);
            // if(pointUpdatedByThisThread > shareDistInBlockPivotCol[threadIdx.x*getBlockSize + k] + shareDistInBlockPivotRow[k*getBlockSize + threadIdx.y]) {
            //     pointUpdatedByThisThread = shareDistInBlockPivotCol[threadIdx.x*getBlockSize + k] + shareDistInBlockPivotRow[k*getBlockSize + threadIdx.y];
            // }
        }
        graph[pivotColI*V + pivotRowJ] = pointUpdatedByThisThread;
    }
}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW() {
    // malloc space on gpu
    int * deviceGraph;
    deviceGraph = (int *)malloc(padN*padN*sizeof(int));
    hipHostRegister(Dist, padN*padN*sizeof(int), hipHostRegisterDefault);
    hipMalloc(&deviceGraph, padN*padN*sizeof(int));
    hipMemcpy(deviceGraph, Dist,  padN*padN*sizeof(int), hipMemcpyHostToDevice);

    // initial parameter
    int round = ceil(n, 32);
    dim3 gridSize(1, 1);
    dim3 gridSize2(round, 1);
    dim3 gridSize3(round, round);
    dim3 blockSize(32, 32);
    
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<n; j++) {
    //         printf("%d ", Dist[i*n+j]);
    //     }
    //     printf("\n");
    // }

    for (int r = 0; r < round; ++r) {
        printf("%d %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        // calculate diagonal axis block (upper left to down right)
        blockedASAPPhase1<<<gridSize, blockSize>>> (r, deviceGraph, padN, 32);

        /* Phase 2*/
        // calculate horizontal axis block
        // blockedASAPPhase2Row<<<gridSize2, blockSize>>> (r, deviceGraph, n, 32);
        // calculate vertical axis block
        // blockedASAPPhase2Col<<<gridSize2, blockSize>>> (r, deviceGraph, n, 32);
        blockedASAPPhase2<<<gridSize2, blockSize>>> (r, deviceGraph, padN, 32);
        /* Phase 3*/
        blockedASAPPhase3<<<gridSize3, blockSize>>> (r, deviceGraph, padN, 32);
    }
    hipMemcpy(Dist, deviceGraph,  padN*padN*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(deviceGraph);
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<n; j++) {
    //         printf("%d ", Dist[i*n+j]);
    //     }
    //     printf("\n");
    // }
    hipFree(deviceGraph);
}
